#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
 * Implements gpusim::FingerprintDB CUDA enabled similarity
 * scoring
 *
 * Copyright Schrodinger LLC, All Rights Reserved.
 --------------------------------------------------------------------------- */

#include <hip/hip_runtime.h>
#include "fingerprintdb_cuda.h"

#include <iostream>
#include <cmath>

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

using std::vector;
using thrust::device_vector;


namespace gpusim
{

typedef device_vector<int> DFingerprint;

/**
 * @internal
 * Functor used to perform tanimoto similarity on GPGPU via thrust::transform
 */
struct TanimotoFunctor {

    const int* m_ref_fp;
    const int m_fp_intsize;
    const int* m_dbdata;

    TanimotoFunctor(const DFingerprint& ref_fp, int fp_intsize, const device_vector<int>& dbdata) : m_ref_fp(ref_fp.data().get()),m_fp_intsize(fp_intsize),m_dbdata(dbdata.data().get())
        {};

    __device__ float
    operator()(const int& fp_index) const
    {
        int total = 0;
        int common = 0;
        int offset = m_fp_intsize*fp_index;
        for(int i=0; i<m_fp_intsize; i++) {
            const int fp1 = m_ref_fp[i];
            const int fp2 = m_dbdata[offset+i];
            total += __popc(fp1) + __popc(fp2); 
            common += __popc(fp1 & fp2);
        }

        return (float)common / (float)(total-common);
    };
};


class FingerprintDBPriv
{
    public:
        device_vector<int> d_data;

};


FingerprintDB::FingerprintDB(int fp_bitcount, int fp_count, const char* data,
            vector<char*>& smiles_vector,
            std::vector<char*>& ids_vector)
{

    m_priv = new FingerprintDBPriv();
    m_fp_intsize = fp_bitcount / (sizeof(int)*8);  //ASSUMES INT-DIVISIBLE SIZE
    m_count = fp_count;

    m_data_size = m_fp_intsize*m_count;
    const int* int_data = (const int*)data;
    m_data.assign(int_data, int_data+m_data_size);

    // Optimization, take the underlying storage of the incoming vectors, 
    // which won't be used again in calling code
    m_smiles.swap(smiles_vector);
    m_ids.swap(ids_vector);

}


void FingerprintDB::copyToGPU(size_t memory_max)
{
    std::cerr << m_data_size << "/" << memory_max << std::endl;
    if(m_data_size > memory_max) 
    {
        std::cerr << "Shrinking db to fit in gpu memory" << std::endl;
    } else {
        std::cerr << "entire db fits in memory, not shrinking" << std::endl;
        m_priv->d_data = m_data;
    }
}


Fingerprint FingerprintDB::getFingerprint(unsigned int index) const
{
    Fingerprint output(m_fp_intsize);

    unsigned int offset = index*m_fp_intsize;
    for(int i=0; i<m_fp_intsize; i++) {
        output[i] = m_data[offset+i];
    }

    return output;
}


void FingerprintDB::search (const Fingerprint& query,
        std::vector<char*>& results_smiles,
        std::vector<char*>& results_ids,
        std::vector<float>& results_scores, unsigned int return_count) const
{
    device_vector<int> d_results_indices(count());
    device_vector<float> d_results_scores(count());

    try
    {
    // Fill indices [0->N), which will be sorted along with scores at end
    thrust::sequence(d_results_indices.begin(), d_results_indices.end());

    // Copy the query fingerprint up to the GPU
    DFingerprint d_ref_fp = query;

    // Use Tanimoto to score similarity of all compounds to query fingerprint
    thrust::transform(d_results_indices.begin(), d_results_indices.end(),
            d_results_scores.begin(),
            TanimotoFunctor(d_ref_fp, m_fp_intsize, m_priv->d_data));

    // Sort scores & indices vectors descending on score
    thrust::sort_by_key(d_results_scores.begin(), d_results_scores.end(),
            d_results_indices.begin(), thrust::greater<float>());
    } catch(thrust::system_error e) {
        std::cerr << "Error!  " << e.what() << std::endl;
    }

    // Push top return_count results to CPU results vectors to be returned
    for(unsigned int i=0;i<return_count;i++) {
        results_smiles.push_back(m_smiles[d_results_indices[i]]);
        results_ids.push_back(m_ids[d_results_indices[i]]);
    }
    results_scores.assign(d_results_scores.begin(),
            d_results_scores.begin()+return_count);

}

/**
 * @brief
 * A CPU implementation of tanimoto similarity, meant purely for testing.
 */
float FingerprintDB::tanimoto_similarity_cpu(const Fingerprint& fp1,
        const Fingerprint& fp2) const
{

    int total = 0;
    int common = 0;
    for(int i=0; i<m_fp_intsize; i++) {
        total += __builtin_popcount(fp1[i]) + __builtin_popcount(fp2[i]); 
        common += __builtin_popcount(fp1[i] & fp2[i]);
    }

    return (float)common / (float)(total-common);
}

size_t get_available_gpu_memory()
{
    size_t free=0, total=0;
    hipMemGetInfo(&free, &total);

    return free;
}

} // namespace gpusim
