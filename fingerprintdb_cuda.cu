#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
 * Implements gpusim::FingerprintDB CUDA enabled similarity
 * scoring
 *
 * Copyright Schrodinger LLC, All Rights Reserved.
 --------------------------------------------------------------------------- */

#include "fingerprintdb_cuda.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <iostream>

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <QDebug>
#include <QFuture>
#include <QMutex>
#include <QtConcurrent/QtConcurrentRun>

using std::make_shared;
using std::pair;
using std::vector;
using thrust::device_vector;

namespace gpusim
{

size_t get_gpu_free_memory(unsigned int device_index)
{
    hipSetDevice(device_index);
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return free;
}

unsigned int get_gpu_count()
{
    static int device_count = 0;
    static bool initialized = false;

    if (!initialized) {
        hipGetDeviceCount(&device_count);
        initialized = true;
    }

    return device_count;
}

unsigned int get_next_gpu(size_t required_memory)
{
    static int next_device = 0;
    for (unsigned int i = 0; i < get_gpu_count(); i++) {
        int gpu =
            next_device++ % get_gpu_count(); // Divide by 0 if called w/o GPU
        auto free = get_gpu_free_memory(i);
        if (free > required_memory) {
            return gpu;
        }
    }
    throw std::runtime_error(
        "Can't find a GPU with enough memory to copy data.");
    return 0; // Never gets here, just for compiler happiness
}

typedef device_vector<int> DFingerprint;

/**
 * @internal
 * Functor used to perform tanimoto similarity on GPGPU via thrust::transform
 */
struct TanimotoFunctor {

    const int* m_ref_fp;
    const int m_fp_intsize;
    const int* m_dbdata;
    const float m_similarity_cutoff;

    TanimotoFunctor(const DFingerprint& ref_fp, int fp_intsize,
                    const device_vector<int>& dbdata, float similarity_cutoff)
        : m_ref_fp(ref_fp.data().get()), m_fp_intsize(fp_intsize),
          m_dbdata(dbdata.data().get()),
          m_similarity_cutoff(similarity_cutoff){};

    __device__ float operator()(const int& fp_index) const
    {
        int total = 0;
        int common = 0;
        int offset = m_fp_intsize * fp_index;
        for (int i = 0; i < m_fp_intsize; i++) {
            const int fp1 = m_ref_fp[i];
            const int fp2 = m_dbdata[offset + i];
            total += __popc(fp1) + __popc(fp2);
            common += __popc(fp1 & fp2);
        }
        float score =
            static_cast<float>(common) / static_cast<float>(total - common);
        return score >= m_similarity_cutoff ? score : 0;
    };
};

struct StorageResultObject {
    vector<SortableResult> m_result_data;
    vector<int> m_approximate_matching_results;
};

class FingerprintDBPriv
{
  public:
    std::shared_ptr<device_vector<int>> d_data;
};

FingerprintDBStorage::FingerprintDBStorage(FingerprintDB* parent,
                                           std::vector<char>& fp_data,
                                           int index_offset, int fp_bitcount)
    : m_parent(parent), m_index_offset(index_offset),
      m_count(fp_data.size() / (fp_bitcount / CHAR_BIT))
{
    const int* int_data = reinterpret_cast<const int*>(fp_data.data());
    const size_t int_size = fp_data.size() / sizeof(int);
    m_data.assign(int_data, int_data + int_size);
}

unsigned int FingerprintDBStorage::getOffsetIndex(unsigned int without_offset)
{
    return without_offset + m_index_offset;
}

FingerprintDB::FingerprintDB(int fp_bitcount, int fp_count,
                             const QString& dbkey, vector<vector<char>>& data,
                             vector<char*>& smiles_vector,
                             std::vector<char*>& ids_vector)
    : m_dbkey(dbkey)
{

    m_fp_intsize = fp_bitcount / (sizeof(int) * 8); // ASSUMES INT-DIVISIBLE
                                                    // SIZE
    m_total_count = fp_count;

    int current_fp_count = 0;
    for (auto& dataset : data) {
        auto storage = make_shared<FingerprintDBStorage>(
            this, dataset, current_fp_count, fp_bitcount);
        storage->m_priv = make_shared<FingerprintDBPriv>();
        m_storage.push_back(storage);
        current_fp_count += storage->m_data.size() / m_fp_intsize;
    }

    if (current_fp_count != m_total_count) {
        throw std::runtime_error("Mismatch between FP count and data, "
                                 "potential database corruption.");
    }

    m_total_data_size = static_cast<size_t>(m_total_count) *
                        static_cast<size_t>(m_fp_intsize) * sizeof(int);
    qDebug() << "Database loaded with" << m_total_count << "molecules";

    // Optimization, take the underlying storage of the incoming vectors,
    // which won't be used again in calling code
    m_smiles.swap(smiles_vector);
    m_ids.swap(ids_vector);
}

void FingerprintDB::copyToGPU(unsigned int fold_factor)
{
    m_fold_factor = fold_factor;
    while (m_fp_intsize % m_fold_factor != 0) {
        m_fold_factor++;
    }

    if (m_fold_factor == 1) {
        for (const auto& storage : m_storage) {
            storage->m_gpu_device =
                get_next_gpu(storage->m_data.size() * sizeof(int));
            hipSetDevice(storage->m_gpu_device);
            // Have to create vector where correct cuda device is set
            storage->m_priv->d_data = make_shared<device_vector<int>>();
            *(storage->m_priv->d_data) = storage->m_data;
        }
    } else {
        for (const auto& storage : m_storage) {
            auto folded_data = fold_data(storage->m_data);
            storage->m_gpu_device =
                get_next_gpu(folded_data.size() * sizeof(int));
            hipSetDevice(storage->m_gpu_device);
            // Have to create vector where correct cuda device is set
            storage->m_priv->d_data = make_shared<device_vector<int>>();
            *(storage->m_priv->d_data) = folded_data;
        }
    }
}

void FingerprintDB::getStorageAndLocalIndex(unsigned int offset_index,
                                            FingerprintDBStorage** storage,
                                            unsigned int* local_index) const
{
    int slice_index_offset = 0;
    *storage = m_storage[0].get();
    for (unsigned int i = 1; i < m_storage.size(); i++) {
        if (m_storage[i]->m_index_offset >= offset_index)
            break;
        *storage = m_storage[i].get();
        slice_index_offset = (*storage)->m_index_offset;
    }
    *local_index = offset_index - slice_index_offset;
}

Fingerprint FingerprintDB::getFingerprint(unsigned int index) const
{
    Fingerprint output(m_fp_intsize);

    FingerprintDBStorage* storage;
    unsigned int local_index;
    getStorageAndLocalIndex(index, &storage, &local_index);

    unsigned int offset = local_index * m_fp_intsize;
    for (int i = 0; i < m_fp_intsize; i++) {
        output[i] = storage->m_data[offset + i];
    }

    return output;
}

void FingerprintDB::search_storage(
    const Fingerprint& query,
    const std::shared_ptr<FingerprintDBStorage>& storage,
    StorageResultObject* results, unsigned int max_return_count,
    float similarity_cutoff) const
{
    auto& sortable_results = results->m_result_data;
    hipSetDevice(storage->m_gpu_device);
    static QMutex mutex;
    vector<int> indices;
    std::vector<char*> results_smiles;
    std::vector<char*> results_ids;
    std::vector<float> results_scores;
    device_vector<float> d_results_scores(storage->m_count);
    device_vector<int> d_results_indices(storage->m_count);
    try {
        // Fill indices [0->N), which will be sorted along with scores at end
        thrust::sequence(d_results_indices.begin(), d_results_indices.end());
        DFingerprint d_ref_fp;
        if (m_fold_factor == 1) {
            // Copy the query fingerprint up to the GPU
            d_ref_fp = query;
        } else {
            auto folded = fold_data(query);
            d_ref_fp = folded;
        }

        const int folded_fp_intsize = m_fp_intsize / m_fold_factor;
        // Use Tanimoto to score similarity of all compounds to query
        // fingerprint
        thrust::transform(d_results_indices.begin(), d_results_indices.end(),
                          d_results_scores.begin(),
                          TanimotoFunctor(d_ref_fp, folded_fp_intsize,
                                          *(storage->m_priv->d_data),
                                          similarity_cutoff));
        auto indices_end = d_results_indices.end();
        auto scores_end = d_results_scores.end();
        if (similarity_cutoff > 0) {
            indices_end = thrust::remove_if(
                d_results_indices.begin(), d_results_indices.end(),
                d_results_scores.begin(), thrust::logical_not<bool>());
            scores_end = thrust::remove(d_results_scores.begin(),
                                        d_results_scores.end(), 0);
        }
        unsigned int indices_size =
            std::distance(d_results_indices.begin(), indices_end);

        mutex.lock();
        results->m_approximate_matching_results.push_back(indices_size);
        mutex.unlock();

        // Sort scores & indices vectors descending on score
        thrust::sort_by_key(d_results_scores.begin(), scores_end,
                            d_results_indices.begin(),
                            thrust::greater<float>());

        int results_to_consider = 0;
        results_to_consider = std::min(
            indices_size, max_return_count * m_fold_factor *
                              static_cast<int>(std::log2(2 * m_fold_factor)));

        indices.assign(d_results_indices.begin(),
                       d_results_indices.begin() + results_to_consider);

    } catch (thrust::system_error e) {
        qDebug() << "Error!" << e.what();
        throw;
    }

    if (m_fold_factor == 1) { // If we don't fold, we can take exact GPU results
        // Push top max_return_count results to CPU results vectors to be
        // returned
        for (auto index : indices) {
            int offset_index = storage->getOffsetIndex(index);
            results_smiles.push_back(m_smiles[offset_index]);
            results_ids.push_back(m_ids[offset_index]);
        }
        results_scores.assign(d_results_scores.begin(),
                              d_results_scores.begin() + indices.size());
    } else { // If we folded, we need to recalculate scores with full
             // fingerprints
        results_scores.resize(indices.size());
        for (unsigned int i = 0; i < indices.size(); i++) {
            int offset_index = storage->getOffsetIndex(indices[i]);
            results_scores[i] =
                tanimoto_similarity_cpu(query, getFingerprint(offset_index));
            // Uncomment below to debug pre vs post folding scores
            // qDebug() << results_scores[i] << " vs " << d_results_scores[i];
        }
        top_results_bubble_sort(indices, results_scores, max_return_count);

        max_return_count = std::min((size_t) max_return_count, indices.size());
        results_scores.resize(max_return_count);
        for (unsigned int i = 0; i < max_return_count; i++) {
            // Check whether the re-scored similarity is too low
            if (results_scores[i] < similarity_cutoff) {
                results_scores.resize(i);
                break;
            }
            results_ids.push_back(m_ids[storage->getOffsetIndex(indices[i])]);
            results_smiles.push_back(
                m_smiles[storage->getOffsetIndex(indices[i])]);
        }
    }

    mutex.lock();
    for (unsigned int i = 0; i < results_smiles.size(); i++) {
        sortable_results.push_back(SortableResult(
            results_scores[i], ResultData(results_smiles[i], results_ids[i])));
    }
    mutex.unlock();
}

void FingerprintDB::search(const Fingerprint& query, const QString& dbkey,
                           unsigned int max_return_count,
                           float similarity_cutoff,
                           std::vector<char*>& results_smiles,
                           std::vector<char*>& results_ids,
                           std::vector<float>& results_scores,
                           unsigned long& approximate_result_count) const
{
    if (dbkey != m_dbkey) {
        qDebug() << "Key check failed, returning empty results";
        return;
    }
    StorageResultObject results;
    auto& sortable_results = results.m_result_data;

    vector<QFuture<void>> futures;
    for (auto& storage : m_storage) {
        QFuture<void> future = QtConcurrent::run(
            this, &FingerprintDB::search_storage, query, storage, &results,
            max_return_count, similarity_cutoff);
        futures.push_back(future);
    }
    for (auto& future : futures) {
        future.waitForFinished();
    }
    std::sort(sortable_results.rbegin(), sortable_results.rend());
    approximate_result_count =
        std::accumulate(results.m_approximate_matching_results.begin(),
                        results.m_approximate_matching_results.end(), 0);

    for (auto result : sortable_results) {
        results_scores.push_back(result.first);
        results_smiles.push_back(result.second.first);
        results_ids.push_back(result.second.second);
    }
    int result_size = std::min(static_cast<int>(max_return_count),
                               static_cast<int>(results_scores.size()));
    results_scores.resize(result_size);
    results_smiles.resize(result_size);
    results_ids.resize(result_size);
}

/**
 * @brief
 * A CPU implementation of tanimoto similarity, meant purely for testing.
 */
float FingerprintDB::tanimoto_similarity_cpu(const Fingerprint& fp1,
                                             const Fingerprint& fp2) const
{

    int total = 0;
    int common = 0;
    for (int i = 0; i < m_fp_intsize; i++) {
        total += __builtin_popcount(fp1[i]) + __builtin_popcount(fp2[i]);
        common += __builtin_popcount(fp1[i] & fp2[i]);
    }

    return (float) common / (float) (total - common);
}

size_t get_available_gpu_memory()
{
    size_t free = 0;
    for (unsigned int gpu = 0; gpu < get_gpu_count(); gpu++) {
        auto lfree = get_gpu_free_memory(gpu);
        free += lfree;
    }

    // Comment out below line to force-test folding:
    // free = 100*1024*1024;

    return free;
}

} // namespace gpusim
