#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
 * Implements gpusim::FingerprintDB CUDA enabled similarity
 * scoring
 *
 * Copyright Schrodinger LLC, All Rights Reserved.
 --------------------------------------------------------------------------- */

#include "fingerprintdb_cuda.h"

#include <iostream>
#include <cmath>

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

using std::vector;
using thrust::device_vector;


namespace gpusim
{

typedef device_vector<int> DFingerprint;

/**
 * @internal
 * Functor used to perform tanimoto similarity on GPGPU via thrust::transform
 */
struct TanimotoFunctor {

    const int* m_ref_fp;
    const int m_fp_intsize;
    const int* m_dbdata;

    TanimotoFunctor(const DFingerprint& ref_fp, int fp_intsize, const device_vector<int>& dbdata) : m_ref_fp(ref_fp.data().get()),m_fp_intsize(fp_intsize),m_dbdata(dbdata.data().get())
        {};

    __device__ float
    operator()(const int& fp_index) const
    {
        int total = 0;
        int common = 0;
        int offset = m_fp_intsize*fp_index;
        for(int i=0; i<m_fp_intsize; i++) {
            const int fp1 = m_ref_fp[i];
            const int fp2 = m_dbdata[offset+i];
            total += __popc(fp1) + __popc(fp2); 
            common += __popc(fp1 & fp2);
        }

        return (float)common / (float)(total-common);
    };
};


/**
 * @internal
 * Functor used to perform tanimoto similarity on CPU via std::transform
 */
class TanimotoFunctorCPU{
    public:

    const int* m_ref_fp;
    const int m_fp_intsize;
    const int* m_dbdata;

    TanimotoFunctorCPU(const Fingerprint& ref_fp, int fp_intsize, const vector<int>& dbdata) : m_ref_fp(ref_fp.data()),m_fp_intsize(fp_intsize),m_dbdata(dbdata.data())
        {};

    float operator()(const int& fp_index) const
    {
        int total = 0;
        int common = 0;
        int offset = m_fp_intsize*fp_index;
        for(int i=0; i<m_fp_intsize; i++) {
            const int fp1 = m_ref_fp[i];
            const int fp2 = m_dbdata[offset+i];
            total += __builtin_popcount(fp1) + __builtin_popcount(fp2); 
            common += __builtin_popcount(fp1 & fp2);
        }

        return (float)common / (float)(total-common);
    };
};


static inline void swap(vector<int>& indices, vector<float>& scores,
        const int idx1, const int idx2)
{
    int temp = indices[idx1];
    indices[idx1] = indices[idx2];
    indices[idx2] = temp;

    float tempf = scores[idx1];
    scores[idx1] = scores[idx2];
    scores[idx2] = tempf;
}


/**
 * @internal
 * This performs a partial bubble sort, concluding after the top N scores
 * have been sorted.
 * NOTE:  Resulting vectors are *UNSORTED* beyond N positions
 * This version of bubble sort is only O(N*len(scores)), where N is small
 */
void top_results_bubble_sort(vector<int>& indices, vector<float>& scores,
        int number_required)
{
    const int count = indices.size();
    for(int i=0; i<number_required; i++) {
        for(int j=0; j<(count-i-1); j++) {
            if(scores[j] > scores[j+1]) {
                swap(indices, scores, j, j+1);
            }
        }
    }
}


class FingerprintDBPriv
{
    public:
        std::vector<int> data;
        device_vector<int> d_data;

};


FingerprintDB::FingerprintDB(int fp_bitcount, int fp_count, const char* data,
            vector<char*>& smiles_vector,
            std::vector<char*>& ids_vector)
{
    m_priv = new FingerprintDBPriv();
    m_fp_intsize = fp_bitcount / (sizeof(int)*8);  //ASSUMES INT-DIVISIBLE SIZE
    m_count = fp_count;

    int data_size = (fp_bitcount/(sizeof(int)*8))*m_count;
    const int* int_data = (const int*)data;
    m_priv->data.assign(int_data, int_data+data_size);
    m_priv->d_data = m_priv->data;

    // Optimization, take the underlying storage of the incoming vectors, 
    // which won't be used again in calling code
    m_smiles.swap(smiles_vector);
    m_ids.swap(ids_vector);

}


Fingerprint FingerprintDB::getFingerprint(unsigned int index) const
{
    Fingerprint output(m_fp_intsize);

    unsigned int offset = index*m_fp_intsize;
    for(int i=0; i<m_fp_intsize; i++) {
        output[i] = m_priv->data[offset+i];
    }

    return output;
}


void FingerprintDB::search (const Fingerprint& query,
        std::vector<char*>& results_smiles,
        std::vector<char*>& results_ids,
        std::vector<float>& results_scores, unsigned int return_count) const
{
    device_vector<int> d_results_indices(count());
    device_vector<float> d_results_scores(count());

    try
    {
    // Fill indices [0->N), which will be sorted along with scores at end
    thrust::sequence(d_results_indices.begin(), d_results_indices.end());

    // Copy the query fingerprint up to the GPU
    DFingerprint d_ref_fp = query;

    // Use Tanimoto to score similarity of all compounds to query fingerprint
    thrust::transform(d_results_indices.begin(), d_results_indices.end(),
            d_results_scores.begin(),
            TanimotoFunctor(d_ref_fp, m_fp_intsize, m_priv->d_data));

    // Sort scores & indices vectors descending on score
    thrust::sort_by_key(d_results_scores.begin(), d_results_scores.end(),
            d_results_indices.begin(), thrust::greater<float>());
    } catch(thrust::system_error e) {
        std::cerr << "Error!  " << e.what() << std::endl;
    }

    // Push top return_count results to CPU results vectors to be returned
    for(unsigned int i=0;i<return_count;i++) {
        results_smiles.push_back(m_smiles[d_results_indices[i]]);
        results_ids.push_back(m_ids[d_results_indices[i]]);
    }
    results_scores.assign(d_results_scores.begin(),
            d_results_scores.begin()+return_count);

}

void FingerprintDB::search_cpu (const Fingerprint& query,
        std::vector<char*>& results_smiles,
        std::vector<char*>& results_ids,
        std::vector<float>& results_scores, unsigned int return_count) const
{
    const int total = count();
    vector<int> indices(total);
    vector<float> scores(total);

    for(int i=0; i<total; i++) {
        indices[i] = i;
    }

    // Use Tanimoto to score similarity of all compounds to query fingerprint
    std::transform(indices.begin(), indices.end(), scores.begin(),
            TanimotoFunctorCPU(query, m_fp_intsize, m_priv->data));

    top_results_bubble_sort(indices, scores, return_count);

    // Push top return_count results to CPU results vectors to be returned
    for(unsigned int i=total-1;i>=total-return_count;i--) {
        results_smiles.push_back(m_smiles[indices[i]]);
        results_ids.push_back(m_ids[indices[i]]);
        results_scores.push_back(scores[i]);
    }
}

/**
 * @brief
 * A CPU implementation of tanimoto similarity, meant purely for testing.
 */
float FingerprintDB::tanimoto_similarity_cpu(const Fingerprint& fp1,
        const Fingerprint& fp2) const
{

    int total = 0;
    int common = 0;
    for(int i=0; i<m_fp_intsize; i++) {
        total += __builtin_popcount(fp1[i]) + __builtin_popcount(fp2[i]); 
        common += __builtin_popcount(fp1[i] & fp2[i]);
    }

    return (float)common / (float)(total-common);
}

std::vector<int> fold_fingerprint(std::vector<int> &fp, const int factor)
{
    vector<int> new_fp(fp.size()/factor);
    const int INT_SIZE = sizeof(int) * 8;
    const int original_size = INT_SIZE * fp.size();
    // Make sure the new_size is always int-sized
    const int new_size = INT_SIZE * (fp.size() / factor);
    // resize here
    for(int pos=0; pos < original_size; pos++) {
        int int_offset = pos / INT_SIZE;
        int inner_pos = pos % INT_SIZE;
        int bit_on = (fp[int_offset] & (0x01 << inner_pos)) ? 1 : 0;

        int new_pos = pos % new_size;
        int new_int_offset = new_pos / INT_SIZE;
        int new_inner_pos = new_pos % INT_SIZE;
        new_fp[new_int_offset] |= (1 << new_inner_pos) * bit_on;
    }

    return new_fp;
}

} // namespace gpusim
