#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
 * Implements gpusim::FingerprintDB CUDA enabled similarity
 * scoring
 *
 * Copyright Schrodinger LLC, All Rights Reserved.
 --------------------------------------------------------------------------- */

#include <hip/hip_runtime.h>
#include "fingerprintdb_cuda.h"

#include <iostream>
#include <cmath>

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

using std::cerr;
using std::endl;
using std::vector;
using thrust::device_vector;


namespace gpusim
{

typedef device_vector<int> DFingerprint;

/**
 * @internal
 * Functor used to perform tanimoto similarity on GPGPU via thrust::transform
 */
struct TanimotoFunctor {

    const int* m_ref_fp;
    const int m_fp_intsize;
    const int* m_dbdata;
    const float m_similarity_cutoff;

    TanimotoFunctor(const DFingerprint& ref_fp, int fp_intsize,
            const device_vector<int>& dbdata, float similarity_cutoff) :
        m_ref_fp(ref_fp.data().get()),m_fp_intsize(fp_intsize),m_dbdata(dbdata.data().get()),
        m_similarity_cutoff(similarity_cutoff)
        {};

    __device__ float
    operator()(const int& fp_index) const
    {
        int total = 0;
        int common = 0;
        int offset = m_fp_intsize*fp_index;
        for(int i=0; i<m_fp_intsize; i++) {
            const int fp1 = m_ref_fp[i];
            const int fp2 = m_dbdata[offset+i];
            total += __popc(fp1) + __popc(fp2); 
            common += __popc(fp1 & fp2);
        }
        float score = static_cast<float>(common) / static_cast<float>(total-common);
        return score >= m_similarity_cutoff ? score : 0;
    };
};


class FingerprintDBPriv
{
    public:
        device_vector<int> d_data;

};


FingerprintDB::FingerprintDB(int fp_bitcount, int fp_count, const char* data,
            vector<char*>& smiles_vector,
            std::vector<char*>& ids_vector)
{

    m_priv = std::make_shared<FingerprintDBPriv>();
    m_fp_intsize = fp_bitcount / (sizeof(int)*8);  //ASSUMES INT-DIVISIBLE SIZE
    m_count = fp_count;

    m_data_size = m_fp_intsize*m_count;
    const int* int_data = (const int*)data;
    m_data.assign(int_data, int_data+m_data_size);

    // Optimization, take the underlying storage of the incoming vectors, 
    // which won't be used again in calling code
    m_smiles.swap(smiles_vector);
    m_ids.swap(ids_vector);

}


void FingerprintDB::copyToGPU(unsigned int fold_factor)
{
    m_fold_factor = fold_factor;
    while(m_fp_intsize % m_fold_factor != 0) {
        m_fold_factor++;
    }

    if(m_fold_factor == 1) {
        m_priv->d_data = m_data;
    } else {
        m_folded_data = fold_data(m_data);
        m_priv->d_data = m_folded_data;
    }
}


Fingerprint FingerprintDB::getFingerprint(unsigned int index) const
{
    Fingerprint output(m_fp_intsize);

    unsigned int offset = index*m_fp_intsize;
    for(int i=0; i<m_fp_intsize; i++) {
        output[i] = m_data[offset+i];
    }

    return output;
}


void FingerprintDB::search(const Fingerprint& query,
        std::vector<char*>& results_smiles,
        std::vector<char*>& results_ids,
        std::vector<float>& results_scores,
        unsigned int return_count,
        float similarity_cutoff) const
{
    device_vector<int> d_results_indices(count());
    device_vector<float> d_results_scores(count());
    vector<int> indices;
    int results_to_consider = 0;

    try
    {
    // Fill indices [0->N), which will be sorted along with scores at end
    thrust::sequence(d_results_indices.begin(), d_results_indices.end());

    DFingerprint d_ref_fp;
    if(m_fold_factor == 1) {
        // Copy the query fingerprint up to the GPU
        d_ref_fp = query;
    } else {
        auto folded = fold_data(query);
        d_ref_fp = folded;
    }

    int folded_fp_intsize = m_fp_intsize / m_fold_factor;
    // Use Tanimoto to score similarity of all compounds to query fingerprint
    thrust::transform(d_results_indices.begin(), d_results_indices.end(),
            d_results_scores.begin(),
            TanimotoFunctor(d_ref_fp, folded_fp_intsize, m_priv->d_data,
                similarity_cutoff));

    auto indices_end = d_results_indices.end();
    auto scores_end = d_results_scores.end();
    if(similarity_cutoff > 0) {
        indices_end = thrust::remove_if(d_results_indices.begin(),
                d_results_indices.end(), d_results_scores.begin(),
                thrust::logical_not<bool>());
        scores_end = thrust::remove(d_results_scores.begin(),
                d_results_scores.end(), 0);
    }
    unsigned int indices_size = std::distance(d_results_indices.begin(),
            indices_end);

    // Sort scores & indices vectors descending on score
    thrust::sort_by_key(d_results_scores.begin(), scores_end,
            d_results_indices.begin(), thrust::greater<float>());

    results_to_consider = std::min(indices_size, return_count*m_fold_factor);

    indices.assign(d_results_indices.begin(), 
            d_results_indices.begin()+results_to_consider);

    } catch(thrust::system_error e) {
        cerr << "Error!  " << e.what() << endl;
    }


    if(m_fold_factor == 1) { // If we don't fold, we can take exact GPU results
        // Push top return_count results to CPU results vectors to be returned
        for(unsigned int i=0;i<results_to_consider;i++) {
            results_smiles.push_back(m_smiles[d_results_indices[i]]);
            results_ids.push_back(m_ids[d_results_indices[i]]);
        }
        results_scores.assign(d_results_scores.begin(),
                d_results_scores.begin()+results_to_consider);
    } else { // If we folded, we need to recalculate scores with full fingerprints
        results_scores.resize(results_to_consider);
        for(unsigned int i=0;i<results_to_consider;i++) {
            results_scores[i] = tanimoto_similarity_cpu(query,
                    getFingerprint(indices[i]));
            // Uncomment below to debug pre vs post folding scores
            // cerr << results_scores[i] << " vs " << d_results_scores[i] << endl;
        }
        top_results_bubble_sort(indices, results_scores, return_count);

        return_count = std::min((size_t)return_count, indices.size());
        results_scores.resize(return_count);
        for(unsigned int i=0;i<return_count;i++) {
            // Check whether the re-scored similarity is too low
            if(results_scores[i] < similarity_cutoff) {
                results_scores.resize(i);
                break;
            }
            results_ids.push_back(m_ids[indices[i]]);
            results_smiles.push_back(m_smiles[indices[i]]);
        }
    }
}

/**
 * @brief
 * A CPU implementation of tanimoto similarity, meant purely for testing.
 */
float FingerprintDB::tanimoto_similarity_cpu(const Fingerprint& fp1,
        const Fingerprint& fp2) const
{

    int total = 0;
    int common = 0;
    for(int i=0; i<m_fp_intsize; i++) {
        total += __builtin_popcount(fp1[i]) + __builtin_popcount(fp2[i]); 
        common += __builtin_popcount(fp1[i] & fp2[i]);
    }

    return (float)common / (float)(total-common);
}

size_t get_available_gpu_memory()
{
    size_t free=0, total=0;
    hipMemGetInfo(&free, &total);
    // Comment out below line to force-test folding:
    // free = 100*1024*1024;

    return free;
}

} // namespace gpusim
